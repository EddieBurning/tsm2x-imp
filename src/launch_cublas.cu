/**
 * CUBLAS wrapper for different matrix types
 * by Cody Rivera, 2019-2020
 */

#include "hipblas.h"
#include "launch_cublas.cuh"


// float specialization
template <>
hipblasStatus_t launchCublas(hipblasHandle_t handle, float& one, float& zero,
                            const float* devA, const float* devB, float* devC,
                            const unsigned int m, const unsigned int n, 
                            const unsigned int k) {
    return hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &one,
                        devA, HIP_R_32F, m, devB, HIP_R_32F, k, &zero,
                        devC, HIP_R_32F, m, HIP_R_32F,
                        HIPBLAS_GEMM_DEFAULT);
}

// double specialization
template <>
hipblasStatus_t launchCublas(hipblasHandle_t handle, double& one, double& zero,
                            const double* devA, const double* devB, double* devC,
                            const unsigned int m, const unsigned int n, 
                            const unsigned int k) {
    return hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &one,
                        devA, HIP_R_64F, m, devB, HIP_R_64F, k, &zero,
                        devC, HIP_R_64F, m, HIP_R_64F,
                        HIPBLAS_GEMM_DEFAULT);
}
 