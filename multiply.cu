#include "hip/hip_runtime.h"
/*
  multiply.cu -- Matrix multiplication testbench - by Cody Rivera
*/

#include <cstdio>
#include <cmath>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "multiply.cuh"

#include "kernels.cuh"


#define EPS 10e-3
#define MAX_TILES 255

/*
  Helper functions
 */

#define FLOAT_T1 128
#define FLOAT_T2 4
#define FLOAT_T3 4

#define DOUBLE_T1 128
#define DOUBLE_T2 4
#define DOUBLE_T3 4


// Based on https://randomascii.wordpress.com/2012/02/25/comparing-floating-point-numbers-2012-edition/
bool approxEqual(double A, double B,
                 double maxRelDiff = EPS)
{
    // Calculate the difference.
    double diff = fabs(A - B);
    A = fabs(A);
    B = fabs(B);
    // Find the largest
    double largest = (B > A) ? B : A;

    if (diff <= largest * maxRelDiff)
        return true;
    return false;
}


template<typename FloatType>
bool matrixCompare(const FloatType* A, const FloatType* B,
                   unsigned int m, unsigned int n,
                   unsigned int& iFail, unsigned int& jFail)
{
    FloatType aVal, bVal;
    bool b = true;
    // Cache-friendly comparison pattern
    for (unsigned int j = 0; j < n && b; j++)
    {
        for (unsigned int i = 0; i < m && b; i++)
        {
            aVal = A[i + (j * m)];
            bVal = B[i + (j * m)];
            if (!approxEqual(aVal, bVal, EPS))
            {
                iFail = i;
                jFail = j;
                b = false;
            }
        }
    }
    return b;
}

template<typename FloatType>
void reportTestSuccess(const char* testName, double GFLOPs, double totalGFLOPs)
{
    printf("%s succeeded: %g GFLOPs, %g GFLOPs acc. for transfers\n", testName, GFLOPs, totalGFLOPs);
}

template<typename FloatType>
void reportTestFailure(const char* testName,
                       const FloatType* orig, const FloatType* cand,
                       unsigned int leadDim,
                       unsigned int iFail, unsigned int jFail)
{
    double oVal = (double)orig[iFail + (jFail * leadDim)];
    double cVal = (double)cand[iFail + (jFail * leadDim)];
    fprintf(stderr, "%s failed: Original[%u, %u] = %.6f != Candidate[%u, %u] = %.6f\n",
            testName, iFail, jFail, oVal, iFail, jFail, cVal);
}

template<typename FloatType>
double getGFLOPs(double time, unsigned int m, unsigned int n, unsigned int k)
{
    double instCount = ((double) m * (double) n * (double) k) / 10e9;
    double timeSeconds = time / 1000;
    return instCount / timeSeconds;
}




/*
  Executes the kernels
 */
template<>
bool runKernels(const float* A, const float* B, float* C,
                const unsigned int m, const unsigned int n,
                const unsigned int k)
{
    // Candidate for C -- Used by GPU kernels
    float* candC;
    // Device memory
    float* devA, * devB, * devC;
    // Events used for timing
    hipEvent_t start, end, startTotal, endTotal;
    float time, timeTotal;

    printf("Multiplying matrix A[%u, %u] by matrix B[%u, %u]\n\n", m, k, k, n); 

    // Allocates new memory
    candC = (float*)malloc(m * n * sizeof(float));
    if (candC == NULL)
    {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }
    
    cudaErrchk(hipMalloc((float**)&devA, m * k * sizeof(float)));
    cudaErrchk(hipMalloc((float**)&devB, k * n * sizeof(float)));
    cudaErrchk(hipMalloc((float**)&devC, m * n * sizeof(float)));
    
    
    // Inits CUDA events
    cudaErrchk(hipEventCreate(&start));
    cudaErrchk(hipEventCreate(&end));
    cudaErrchk(hipEventCreate(&startTotal));
    cudaErrchk(hipEventCreate(&endTotal));
    
    // Runs CUBLAS call
    hipblasHandle_t handle;
    cublasErrchk(hipblasCreate(&handle));
    
    float one = 1;
    float zero = 0;

    cudaErrchk(hipEventRecord(startTotal));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(float), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(float), hipMemcpyHostToDevice));

    cudaErrchk(hipEventRecord(start));
    cublasErrchk(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              m, n, k,
                              &one, devA, HIP_R_32F, m, devB, HIP_R_32F, k,
                              &zero, devC, HIP_R_32F, m, HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
    cudaErrchk(hipEventRecord(end));
    
    // Copies result back
    cudaErrchk(hipMemcpy(C, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));

    cudaErrchk(hipEventRecord(endTotal));
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
    reportTestSuccess<float>("CUBLAS Test", getGFLOPs<float>(time, m, n, k), getGFLOPs<float>(timeTotal, m, n, k)); 

    cublasErrchk(hipblasDestroy(handle));
    

    // Runs kernels
    // Failure flag
    bool status;
    // Failure indices
    unsigned int iFail, jFail;

    if (m == k)
    {
        // If a TSM

        // Clear result matrix
        cudaErrchk(hipMemset(devC, 0, m * n * sizeof(float)));
        cudaErrchk(hipEventRecord(startTotal));
    
        // Cuda Memory Copy
        cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(float), hipMemcpyHostToDevice));
        cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(float), hipMemcpyHostToDevice));

        int blocks = (k / FLOAT_T1) + 1;
        blocks = (blocks > 65536) ? 65536 : blocks;

        cudaErrchk(hipEventRecord(start));
        floatTSM2Kernel<FLOAT_T1, FLOAT_T2, FLOAT_T3><<<blocks, FLOAT_T1>>>(devA, devB, devC, k, n);
        cudaErrchk(hipGetLastError());
        cudaErrchk(hipEventRecord(end));
    
        // Copies result back
        cudaErrchk(hipMemcpy(candC, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));

        cudaErrchk(hipEventRecord(endTotal));
        cudaErrchk(hipDeviceSynchronize());
        cudaErrchk(hipEventElapsedTime(&time, start, end));
        cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
        status = matrixCompare<float>(C, candC, m, n, iFail, jFail);
        if (status)
        {
            reportTestSuccess<float>("TSM2 Kernel Test", 
                                     getGFLOPs<float>(time, m, n, k), 
                                     getGFLOPs<float>(timeTotal, m, n, k)); 
        }
        else
        {
            reportTestFailure<float>("TSM2 Kernel Test", C, candC, m, iFail, jFail);
        }
        
        
    }

    
    cudaErrchk(hipEventDestroy(start));
    cudaErrchk(hipEventDestroy(end));
    cudaErrchk(hipEventDestroy(startTotal));
    cudaErrchk(hipEventDestroy(endTotal));
    free(candC);
    cudaErrchk(hipFree(devA));
    cudaErrchk(hipFree(devB));
    cudaErrchk(hipFree(devC));
    
    return true;
}




template<>
bool runKernels(const double* A, const double* B, double* C,
                const unsigned int m, const unsigned int n,
                const unsigned int k)
{
    // Candidate for C -- Used by GPU kernels
    double* candC;
    // Device memory
    double* devA, * devB, * devC;
    // Events used for timing
    hipEvent_t start, end, startTotal, endTotal;
    float time, timeTotal;

    printf("Multiplying matrix A[%u, %u] by matrix B[%u, %u]\n\n", m, k, k, n); 

    // Allocates new memory
    candC = (double*)malloc(m * n * sizeof(double));
    if (candC == NULL)
    {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }
    
    cudaErrchk(hipMalloc((double**)&devA, m * k * sizeof(double)));
    cudaErrchk(hipMalloc((double**)&devB, k * n * sizeof(double)));
    cudaErrchk(hipMalloc((double**)&devC, m * n * sizeof(double)));
    
    
    // Inits CUDA events
    cudaErrchk(hipEventCreate(&start));
    cudaErrchk(hipEventCreate(&end));
    cudaErrchk(hipEventCreate(&startTotal));
    cudaErrchk(hipEventCreate(&endTotal));
    
    // Runs CUBLAS call
    hipblasHandle_t handle;
    cublasErrchk(hipblasCreate(&handle));
    
    double one = 1;
    double zero = 0;

    cudaErrchk(hipEventRecord(startTotal));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(double), hipMemcpyHostToDevice));

    cudaErrchk(hipEventRecord(start));
    cublasErrchk(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              m, n, k,
                              &one, devA, HIP_R_64F, m, devB, HIP_R_64F, k,
                              &zero, devC, HIP_R_64F, m, HIP_R_64F, HIPBLAS_GEMM_DEFAULT));
    cudaErrchk(hipEventRecord(end));
    
    // Copies result back
    cudaErrchk(hipMemcpy(C, devC, m * n * sizeof(double), hipMemcpyDeviceToHost));

    cudaErrchk(hipEventRecord(endTotal));
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
    reportTestSuccess<double>("CUBLAS Test", getGFLOPs<double>(time, m, n, k), getGFLOPs<double>(timeTotal, m, n, k)); 

    cublasErrchk(hipblasDestroy(handle));
    

    
    
    // Runs kernel
    // Failure flag
    bool status;
    // Failure indices
    unsigned int iFail, jFail;

    if (m == k)
    {
        // If a TSM

        // Clear result matrix
        cudaErrchk(hipMemset(devC, 0, m * n * sizeof(double)));
        cudaErrchk(hipEventRecord(startTotal));
    
        // Cuda Memory Copy
        cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(double), hipMemcpyHostToDevice));
        cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(double), hipMemcpyHostToDevice));

        int blocks = (k / DOUBLE_T1) + 1;
        blocks = (blocks > 65536) ? 65536 : blocks;

        cudaErrchk(hipEventRecord(start));
        doubleTSM2Kernel<DOUBLE_T1, DOUBLE_T2, DOUBLE_T3><<<blocks, DOUBLE_T1>>>(devA, devB, devC, k, n);
        cudaErrchk(hipGetLastError());
        cudaErrchk(hipEventRecord(end));
    
        // Copies result back
        cudaErrchk(hipMemcpy(candC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost));

        cudaErrchk(hipEventRecord(endTotal));
        cudaErrchk(hipDeviceSynchronize());
        cudaErrchk(hipEventElapsedTime(&time, start, end));
        cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
        status = matrixCompare<double>(C, candC, m, n, iFail, jFail);
        if (status)
        {
            reportTestSuccess<double>("TSM2 Kernel Test", 
                                     getGFLOPs<double>(time, m, n, k), 
                                     getGFLOPs<double>(timeTotal, m, n, k)); 
        }
        else
        {
            reportTestFailure<double>("TSM2 Kernel Test", C, candC, m, iFail, jFail);
        }
        
        
    }



    cudaErrchk(hipEventDestroy(start));
    cudaErrchk(hipEventDestroy(end));
    cudaErrchk(hipEventDestroy(startTotal));
    cudaErrchk(hipEventDestroy(endTotal));
    free(candC);
    cudaErrchk(hipFree(devA));
    cudaErrchk(hipFree(devB));
    cudaErrchk(hipFree(devC));
    
    return true;
}