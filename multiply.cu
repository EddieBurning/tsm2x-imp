#include "hip/hip_runtime.h"
/*
  multiply.cu -- Matrix multiplication testbench - by Cody Rivera
*/

#include <cstdio>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "multiply.cuh"

#define EPS 10e-2
#define MAX_TILES 255

/*
  Helper functions
 */


template<typename FloatType>
bool matrixCompare(const FloatType* A, const FloatType* B,
                   unsigned int m, unsigned int n,
                   unsigned int& iFail, unsigned int& jFail)
{
    FloatType aVal, bVal;
    bool b = true;
    // Cache-friendly comparison pattern
    for (unsigned int j = 0; j < n && b; j++)
    {
        for (unsigned int i = 0; i < m && b; i++)
        {
            aVal = A[i + (j * m)];
            bVal = B[i + (j * m)];
            if (fabs(aVal - bVal) > EPS)
            {
                iFail = i;
                jFail = j;
                b = false;
            }
        }
    }
    return b;
}

template<typename FloatType>
void reportTestSuccess(const char* testName, double GFLOPs, double totalGFLOPs)
{
    printf("%s succeeded: %g GFLOPs, %g GFLOPs acc. for transfers\n", testName, GFLOPs, totalGFLOPs);
}

template<typename FloatType>
void reportTestFailure(const char* testName,
                       const FloatType* orig, const FloatType* cand,
                       unsigned int leadDim,
                       unsigned int iFail, unsigned int jFail)
{
    double oVal = (double)orig[iFail + (jFail * leadDim)];
    double cVal = (double)cand[iFail + (jFail * leadDim)];
    fprintf(stderr, "%s failed: Original[%u, %u] = %.6f != Candidate[%u, %u] = %.6f\n",
            testName, iFail, jFail, oVal, iFail, jFail, cVal);
}

template<typename FloatType>
double getGFLOPs(double time, unsigned int m, unsigned int n, unsigned int k)
{
    double instCount = ((double) m * (double) n * (double) k) / 10e9;
    double timeSeconds = time / 1000;
    return instCount / timeSeconds;
}




/*
  Executes the kernels
 */
template<>
bool runKernels(const float* A, const float* B, float* C,
                const unsigned int m, const unsigned int n,
                const unsigned int k)
{
    // Candidate for C -- Used by GPU kernels
    float* candC;
    // Device memory
    float* devA, * devB, * devC;
    // Events used for timing
    hipEvent_t start, end, startTotal, endTotal;
    float time, timeTotal;

    printf("Multiplying matrix A[%u, %u] by matrix B[%u, %u]\n\n", m, k, k, n); 

    // Allocates new memory
    candC = (float*)malloc(m * n * sizeof(float));
    if (candC == NULL)
    {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }
    
    cudaErrchk(hipMalloc((float**)&devA, m * k * sizeof(float)));
    cudaErrchk(hipMalloc((float**)&devB, k * n * sizeof(float)));
    cudaErrchk(hipMalloc((float**)&devC, m * n * sizeof(float)));
    
    
    // Inits CUDA events
    cudaErrchk(hipEventCreate(&start));
    cudaErrchk(hipEventCreate(&end));
    cudaErrchk(hipEventCreate(&startTotal));
    cudaErrchk(hipEventCreate(&endTotal));
    
    // Runs CUBLAS call
    hipblasHandle_t handle;
    cublasErrchk(hipblasCreate(&handle));
    
    float one = 1;
    float zero = 0;

    cudaErrchk(hipEventRecord(startTotal));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(float), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(float), hipMemcpyHostToDevice));

    cudaErrchk(hipEventRecord(start));
    cublasErrchk(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             m, n, k,
                             &one, devA, m, devB, k,
                             &zero, devC, m));
    cudaErrchk(hipEventRecord(end));
    
    // Copies result back
    cudaErrchk(hipMemcpy(C, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));

    cudaErrchk(hipEventRecord(endTotal));
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
    reportTestSuccess<float>("CUBLAS Test", getGFLOPs<float>(time, m, n, k), getGFLOPs<float>(timeTotal, m, n, k)); 

    cublasErrchk(hipblasDestroy(handle));
    

    /*
    // Runs kernels
    // Failure flag
    bool status;
    // Failure indices
    unsigned int iFail, jFail;
    */

    cudaErrchk(hipEventDestroy(start));
    cudaErrchk(hipEventDestroy(end));
    cudaErrchk(hipEventDestroy(startTotal));
    cudaErrchk(hipEventDestroy(endTotal));
    free(candC);
    cudaErrchk(hipFree(devA));
    cudaErrchk(hipFree(devB));
    cudaErrchk(hipFree(devC));
    
    return true;
}




template<>
bool runKernels(const double* A, const double* B, double* C,
                const unsigned int m, const unsigned int n,
                const unsigned int k)
{
    // Candidate for C -- Used by GPU kernels
    double* candC;
    // Device memory
    double* devA, * devB, * devC;
    // Events used for timing
    hipEvent_t start, end, startTotal, endTotal;
    float time, timeTotal;

    printf("Multiplying matrix A[%u, %u] by matrix B[%u, %u]\n\n", m, k, k, n); 

    // Allocates new memory
    candC = (double*)malloc(m * n * sizeof(double));
    if (candC == NULL)
    {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }
    
    cudaErrchk(hipMalloc((double**)&devA, m * k * sizeof(double)));
    cudaErrchk(hipMalloc((double**)&devB, k * n * sizeof(double)));
    cudaErrchk(hipMalloc((double**)&devC, m * n * sizeof(double)));
    
    
    // Inits CUDA events
    cudaErrchk(hipEventCreate(&start));
    cudaErrchk(hipEventCreate(&end));
    cudaErrchk(hipEventCreate(&startTotal));
    cudaErrchk(hipEventCreate(&endTotal));
    
    // Runs CUBLAS call
    hipblasHandle_t handle;
    cublasErrchk(hipblasCreate(&handle));
    
    double one = 1;
    double zero = 0;

    cudaErrchk(hipEventRecord(startTotal));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(double), hipMemcpyHostToDevice));

    cudaErrchk(hipEventRecord(start));
    cublasErrchk(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             m, n, k,
                             &one, devA, m, devB, k,
                             &zero, devC, m));
    cudaErrchk(hipEventRecord(end));
    
    // Copies result back
    cudaErrchk(hipMemcpy(C, devC, m * n * sizeof(double), hipMemcpyDeviceToHost));

    cudaErrchk(hipEventRecord(endTotal));
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
    reportTestSuccess<double>("CUBLAS Test", getGFLOPs<double>(time, m, n, k), getGFLOPs<double>(timeTotal, m, n, k)); 

    cublasErrchk(hipblasDestroy(handle));
    

    /*
    // Runs kernels
    // Failure flag
    bool status;
    // Failure indices
    unsigned int iFail, jFail;
    */

    cudaErrchk(hipEventDestroy(start));
    cudaErrchk(hipEventDestroy(end));
    cudaErrchk(hipEventDestroy(startTotal));
    cudaErrchk(hipEventDestroy(endTotal));
    free(candC);
    cudaErrchk(hipFree(devA));
    cudaErrchk(hipFree(devB));
    cudaErrchk(hipFree(devC));
    
    return true;
}