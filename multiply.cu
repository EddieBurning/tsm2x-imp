#include "hip/hip_runtime.h"
/*
  multiply.cu -- Matrix multiplication testbench - by Cody Rivera
*/

#include <cstdio>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "multiply.cuh"

#define EPS 10e-2
#define MAX_TILES 255

/*
  Helper functions
 */


bool matrixCompare(const float* A, const float* B,
                   unsigned int m, unsigned int n,
                   unsigned int& iFail, unsigned int& jFail)
{
    float aVal, bVal;
    // Cache-friendly comparison pattern
    for (unsigned int j = 0; j < n; j++)
    {
        for (unsigned int i = 0; i < m; i++)
        {
            aVal = A[i + (j * m)];
            bVal = B[i + (j * m)];
            if (fabs(aVal - bVal) > EPS)
            {
                iFail = i;
                jFail = j;
                return false;
            }
        }
    }
    return true;
}

void reportTestSuccess(const char* testName, double GFLOPs)
{
    printf("%s succeeded: %g GFLOPs\n", testName, GFLOPs);
}

void reportTestFailure(const char* testName,
                       const float* orig, const float* cand,
                       unsigned int leadDim,
                       unsigned int iFail, unsigned int jFail)
{
    double oVal = orig[iFail + (jFail * leadDim)];
    double cVal = cand[iFail + (jFail * leadDim)];
    fprintf(stderr, "%s failed: Original[%u, %u] = %.6f != Candidate[%u, %u] = %.6f\n",
            testName, iFail, jFail, oVal, iFail, jFail, cVal);
}

double getGFLOPs(double time, unsigned int m, unsigned int n, unsigned int k)
{
    double instCount = ((double) m * (double) n * (double) k) / 10e9;
    double timeSeconds = time / 1000;
    return instCount / timeSeconds;
}




/*
  Executes the kernels
 */
bool runKernels(const float* A, const float* B, float* C,
                const unsigned int m, const unsigned int n,
                const unsigned int k)
{
    // Candidate for C -- Used by GPU kernels
    float* candC;
    // Device memory
    float* devA, * devB, * devC;
    // Events used for timing
    hipEvent_t start, end;
    float time;

    printf("Multiplying matrix A[%u, %u] by matrix B[%u, %u]\n\n", m, k, k, n); 

    // Allocates new memory
    candC = (float*)malloc(m * n * sizeof(float));
    if (candC == NULL)
    {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }
    
    cudaErrchk(hipMalloc((float**)&devA, m * k * sizeof(float)));
    cudaErrchk(hipMalloc((float**)&devB, k * n * sizeof(float)));
    cudaErrchk(hipMalloc((float**)&devC, m * n * sizeof(float)));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(float), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(float), hipMemcpyHostToDevice));

    // Inits CUDA events
    cudaErrchk(hipEventCreate(&start));
    cudaErrchk(hipEventCreate(&end));
    
    // Runs CUBLAS call
    hipblasHandle_t handle;
    cublasErrchk(hipblasCreate(&handle));
    
    float one = 1;
    float zero = 0;

    cudaErrchk(hipEventRecord(start));
    cublasErrchk(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             m, n, k,
                             &one, devA, m, devB, k,
                             &zero, devC, m));
    cudaErrchk(hipEventRecord(end));
    
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    
    reportTestSuccess("CUBLAS Test", getGFLOPs(time, m, n, k)); 
    // Copies result back
    cudaErrchk(hipMemcpy(C, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));
    cublasErrchk(hipblasDestroy(handle));
    
    // Runs kernels
    // Failure flag
    bool status;
    // Failure indices
    unsigned int iFail, jFail;
    // Calculates tile numbers
    unsigned int blocksX = (m / TILE_WIDTH) + 1;
    unsigned int blocksY = (n / TILE_WIDTH) + 1;
    if (blocksX > MAX_TILES)
    {
        blocksX = MAX_TILES;
    }
    if (blocksY > MAX_TILES)
    {
        blocksY = MAX_TILES;
    }

    dim3 numBlocks(blocksX, blocksY), blockSize(TILE_WIDTH, TILE_WIDTH);

    // Naive Kernel
    cudaErrchk(hipEventRecord(start));
    naiveGEMMKernel<<<numBlocks, blockSize>>>(devA, devB, devC, m, n, k);
    cudaErrchk(hipEventRecord(end));
    cudaErrchk(hipGetLastError());

    // Timing
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    
    // Copying, checking and reporting
    cudaErrchk(hipMemcpy(candC, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));
    status = matrixCompare(C, candC, m, n, iFail, jFail);
    if (status)
    {
        reportTestSuccess("Naive test", getGFLOPs(time, m, n, k));
    }
    else
    {
        reportTestFailure("Naive test", C, candC, m, iFail, jFail);
        return false;
    }


    // Shared Kernel
    size_t sharedSize = 2 * ((blockSize.x * blockSize.y) + blockSize.y);
    cudaErrchk(hipEventRecord(start));
    sharedGEMMKernel<<<numBlocks, blockSize, sharedSize * sizeof(float)>>>(devA, devB, devC, m, n, k);
    cudaErrchk(hipEventRecord(end));
    cudaErrchk(hipGetLastError());

    // Timing
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    
    // Copying, checking and reporting
    cudaErrchk(hipMemcpy(candC, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));
    status = matrixCompare(C, candC, m, n, iFail, jFail);
    if (status)
    {
        reportTestSuccess("Shared test", getGFLOPs(time, m, n, k));
    }
    else
    {
        reportTestFailure("Shared test", C, candC, m, iFail, jFail);
        return false;
    }

    // Opt Kernel
    cudaErrchk(hipEventRecord(start));
    optGEMMKernel<<<numBlocks, blockSize, sharedSize * sizeof(float)>>>(devA, devB, devC, m, n, k);
    cudaErrchk(hipEventRecord(end));
    cudaErrchk(hipGetLastError());

    // Timing
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    
    // Copying, checking and reporting
    cudaErrchk(hipMemcpy(candC, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));
    status = matrixCompare(C, candC, m, n, iFail, jFail);
    if (status)
    {
        reportTestSuccess("Optimal test", getGFLOPs(time, m, n, k));
    }
    else
    {
        reportTestFailure("Optimal test", C, candC, m, iFail, jFail);
        return false;
    }

    // Deletes memory
    cudaErrchk(hipEventDestroy(start));
    cudaErrchk(hipEventDestroy(end));
    free(candC);
    cudaErrchk(hipFree(devA));
    cudaErrchk(hipFree(devB));
    cudaErrchk(hipFree(devC));
    
    return true;
}